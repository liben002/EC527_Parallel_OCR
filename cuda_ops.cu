#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void CUDA_MAT_SUBT(T *d_A, T *d_B, T *d_C, int row_len, int col_len)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < row_len && col < col_len)
		d_C[row*col_len+col] = d_A[row*col_len+col] - d_B[row*col_len+col];
}

template <typename T>
__global__ void CUDA_MAT_MULT(T* A, T* B, T* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {

    float CValue = 0;

    int Row = blockIdx.y + threadIdx.y;
    int Col = blockIdx.x + threadIdx.x;

    for (int k = 0; k < (ACols); k++) {
            if ((k < ACols && Row < ARows) && (k < BRows && Col < BCols))
                CValue += A[Row*ACols + k] * B[(k)*BCols + Col];

    }

    if (Row < CRows && Col < CCols){
    	C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols)+(blockIdx.x*blockDim.x)+threadIdx.x]=CValue;
    }
}
