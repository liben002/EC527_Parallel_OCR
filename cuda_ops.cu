// CUDA operations fro matrix multiply
// References: 	https://stackoverflow.com/questions/35799478/how-to-implement-a-nxm-cuda-matrix-multiplication
// 				https://stackoverflow.com/questions/18997773/non-square-matrix-multiplication-in-cuda
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define SHARED_TILE_WIDTH 16

template <typename T>
__global__ void CUDA_MAT_MULT_NORMAL(T *d_A, T *d_B, T *d_C, int rows_A, int cols_A, int rows_B, int cols_B, int rows_C, int cols_C)
{

	int row = blockIdx.x * blockDim.x + threadIdx.x; // for d_A matrix
	int col = blockIdx.y * blockDim.y + threadIdx.y; // for d_B matrix

	float c_val = 0;

	if (row < rows_C && col < cols_C) // only want rows and columns that fit within the resultant matrix, otherwise, doing extra work
	{
		for (int i = 0; i < cols_A; i++) {
			if (row < rows_A && (i < rows_B && col < cols_B)) // explicitly check bounds
			{
				__syncthreads();
				c_val += d_A[row * cols_A + i] * d_B[i * cols_B + col];
				__syncthreads();
			}
		}
		d_C[row * cols_C + col] = c_val;
	}
}

template <typename T>
__global__ void CUDA_MAT_MULT_TILED(T *d_A, T *d_B, T *d_C, int rows_A, int cols_A, int rows_B, int cols_B, int rows_C, int cols_C, int TILE_WIDTH)
{

	int row = blockIdx.x * TILE_WIDTH + threadIdx.x; // for d_A matrix
	int col = blockIdx.y * TILE_WIDTH + threadIdx.y; // for d_B matrix

	int c_row = blockIdx.x * blockDim.x + threadIdx.x;
	int c_col = blockIdx.y * blockDim.y + threadIdx.y;

	T c_val = 0;

	if (row < rows_C && col < cols_C) // only want rows and columns that fit within the resultant matrix, otherwise, doing extra work
	{
		for (int i = 0; i < (cols_A + TILE_WIDTH - 1)/TILE_WIDTH; i++)
		{
			for (int j = 0; j < TILE_WIDTH; j++)
			{
				if ((i * TILE_WIDTH + j < cols_A && row < rows_A) && (i * TILE_WIDTH + j < rows_B && col < cols_B)) // don't go overbounds since d_A and d_B are not necessarily the same shape
				{
					c_val += d_A[row * cols_A + i * TILE_WIDTH + j] * d_B[(i * TILE_WIDTH + j) * cols_B + col];
				}
			}
		}

		d_C[(c_row * cols_C) + c_col] = c_val;
	}
}

template <typename T>
__global__ void CUDA_MAT_MULT_SHARED(T *d_A, T *d_B, T *d_C, int rows_A, int cols_A, int rows_B, int cols_B, int rows_C, int cols_C)
{

	int row = blockIdx.x * SHARED_TILE_WIDTH + threadIdx.x; // for d_A matrix
	int col = blockIdx.y * SHARED_TILE_WIDTH + threadIdx.y; // fpr d_B matrix

	int c_row = blockIdx.x * blockDim.x + threadIdx.x;
	int c_col = blockIdx.y * blockDim.y + threadIdx.y;

	T c_val = 0;

	__shared__ T s_A[SHARED_TILE_WIDTH][SHARED_TILE_WIDTH];
	__shared__ T s_B[SHARED_TILE_WIDTH][SHARED_TILE_WIDTH];

	for (int i = 0; i < (SHARED_TILE_WIDTH + cols_A - 1)/SHARED_TILE_WIDTH; i++) {

		s_A[threadIdx.x][threadIdx.y] = (i * SHARED_TILE_WIDTH + threadIdx.y < cols_A && row < rows_A) ? d_A[row * cols_A + i * SHARED_TILE_WIDTH + threadIdx.y] : 0.0;

		s_B[threadIdx.x][threadIdx.y] = (i * SHARED_TILE_WIDTH + threadIdx.x < rows_B && col < cols_B) ? d_B[(i * SHARED_TILE_WIDTH + threadIdx.x) * cols_B + col] : 0.0;

		__syncthreads();
		for (int j = 0; j < SHARED_TILE_WIDTH; j++)
		{
			c_val += s_A[threadIdx.x][j] * s_B[j][threadIdx.y];
		}
		__syncthreads();
	}

	if (row < rows_C && col < cols_C)
	{
		d_C[c_row * cols_C + c_col] = c_val;
	}
}