#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void CUDA_MAT_SUBT(T *d_A, T *d_B, T *d_C, int row_len, int col_len)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < row_len && col < col_len)
		d_C[row*col_len+col] = d_A[row*col_len+col] - d_B[row*col_len+col];
}

template <typename T>
__global__ void CUDA_MAT_MULT(T *d_A, T *d_B, T *d_C, int row_len, int col_len)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	float Pval = 0;

	for (int k = 0; k < row_len; k++) {
		Pval += Md[row*row_len+k] * Nd[k*row_len+col];
		__syncthreads();
	}

	Pd[row*row_len+col] = Pval;
}

