#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void CUDA_MAT_SUBT(T *d_A, T *d_B, T *d_C, int row_len, int col_len)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < row_len && col < col_len)
		d_C[row*col_len+col] = d_A[row*col_len+col] - d_B[row*col_len+col];
}

template <typename T>
__global__ void CUDA_MAT_MULT(T *d_A, T *d_B, T *d_C, int row_len_dA, int col_len_dA, int row_len_dB, int col_len_dB)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	float Pval = 0;

	if (!(row >= row_len_dA || col >= col_len_dA || row >= row_len_dB || col >= col_len_dB))
	{
		for (int k = 0; k < row_len; k++) {
			Pval += d_A[row*row_len_dA+k] * d_B[k*row_len_dB+col];
			__syncthreads();
		}

	d_C[row*row_len+col] = Pval;
}

