// CUDA operations fro matrix multiply
// References: 	https://stackoverflow.com/questions/35799478/how-to-implement-a-nxm-cuda-matrix-multiplication
// 				https://stackoverflow.com/questions/18997773/non-square-matrix-multiplication-in-cuda
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define SHARED_TILE_WIDTH 16

template <typename T>
__global__ void CUDA_MAT_MULT_NORMAL(T *A, T *B, T *C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {

	float CValue = 0;

    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    int Col = blockIdx.x*blockDim.x + threadIdx.x;

    for (int k = 0; k < ACols; k++) {

            if ((k < ACols && Row < ARows) && (k < BRows && Col < BCols))
                CValue += A[Row*ACols + k] * B[(k)*BCols + Col];

    }

    if (Row < CRows && Col < CCols) C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols)+(blockIdx.x*blockDim.x)+threadIdx.x]=CValue;
}

template <typename T>
__global__ void CUDA_MAT_MULT_TILED(T* d_A, T* d_B, T* d_C, int rows_A, int cols_A, int rows_B, int cols_B, int rows_C, int cols_C, int TILE_WIDTH) {

	int row = blockIdx.y * TILE_WIDTH + threadIdx.y; // for d_A matrix
	int col = blockIdx.x * TILE_WIDTH + threadIdx.x; // for d_B matrix

	int c_row = blockIdx.y * blockDim.y + threadIdx.y;
	int c_col = blockIdx.x * blockDim.x + threadIdx.x;

	T c_val = 0;

	if (row < rows_C && col < cols_C) // only want rows and columns that fit within the resultant matrix, otherwise, doing extra work
	{
		for (int i = 0; i < (cols_A + TILE_WIDTH - 1)/TILE_WIDTH; i++)
		{
			for (int k = 0; k < TILE_WIDTH; k++)
			{
				if ((i * TILE_WIDTH + k < cols_A && row < rows_A) && (i * TILE_WIDTH + k < rows_B && col < cols_B)) // don't go overbounds since d_A and d_B are not necessarily the same shape
				{
					c_val += d_A[row * cols_A + i * TILE_WIDTH + k] * d_B[(i * TILE_WIDTH + k) * cols_B + col];
				}
			}
		}

		d_C[(c_row * cols_C) + c_col] = c_val;
	}
}

template <typename T>
__global__ void CUDA_MAT_MULT_SHARED_TILED(T *d_A, T *d_B, T *d_C, int rows_A, int cols_A, int rows_B, int cols_B, int rows_C, int cols_C)
{
	T c_val = 0;

	int row = blockIdx.y*SHARED_TILE_WIDTH + threadIdx.y;
	int col = blockIdx.x*SHARED_TILE_WIDTH + threadIdx.x;

	__shared__ T s_A[SHARED_TILE_WIDTH][SHARED_TILE_WIDTH];
	__shared__ T s_B[SHARED_TILE_WIDTH][SHARED_TILE_WIDTH];

	for (int k = 0; k < (SHARED_TILE_WIDTH + cols_A - 1)/SHARED_TILE_WIDTH; k++) {
		if (k*SHARED_TILE_WIDTH + threadIdx.x < cols_A && row < rows_A)
		{
			s_A[threadIdx.y][threadIdx.x] = d_A[row*cols_A + k*SHARED_TILE_WIDTH + threadIdx.x];
		}
		else
		{
			s_A[threadIdx.y][threadIdx.x] = 0.0;
		}

		if (k*SHARED_TILE_WIDTH + threadIdx.y < rows_B && col < cols_B)
		{
			s_B[threadIdx.y][threadIdx.x] = d_B[(k*SHARED_TILE_WIDTH + threadIdx.y)*cols_B + col];
		}
		else
		{
			s_B[threadIdx.y][threadIdx.x] = 0.0;
		}

		__syncthreads();

		for (int n = 0; n < SHARED_TILE_WIDTH; ++n)
			c_val += s_A[threadIdx.y][n] * s_B[n][threadIdx.x];

		__syncthreads();
	}

	if (row < rows_C && col < cols_C)
	{
		d_C[((blockIdx.y * blockDim.y + threadIdx.y)*cols_C) + (blockIdx.x * blockDim.x)+ threadIdx.x] = c_val;
	}
}