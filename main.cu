#include "hip/hip_runtime.h"
#include "nn.hpp"
#include <iostream>
#include <time.h>
#include <cassert>

#define EPOCHS 100
#define START 600
#define END 1600
#define STEP_SIZE 100

/**
 * Function to test neural network
 * @returns none
 */
static void test(int row_length) {
	// Creating network with 3 layers for "iris.csv"
	// First layer neurons must match testing params
	machine_learning::neural_network::NeuralNetwork myNN = machine_learning::neural_network::NeuralNetwork({ {4, "none"}, {row_length, "relu"}, {row_length, "sigmoid"} });

	// Printing summary of model
	myNN.summary();

	// Training Model
	myNN.fit_from_csv("iris.csv", true, EPOCHS, 0.3, false, 2, 32, true);

	printf("Testing predictions\n");
	// Testing predictions of model
	assert(machine_learning::argmax(myNN.single_predict({{5, 3.4, 1.6, 0.4}})) == 0);
	assert(machine_learning::argmax( myNN.single_predict({{6.4, 2.9, 4.3, 1.3}})) == 1);
	assert(machine_learning::argmax(myNN.single_predict({{6.2, 3.4, 5.4, 2.3}})) == 2);
	return;
}

/**
 * @brief Main function
 * @returns 0 on exit
 */
int main() {

	double duration_table[(END-START) / STEP_SIZE + 1][2];

	for (int i = START; i <= END; i+= STEP_SIZE)
	{
		duration_table[i/100 -1][0] = i;
		printf("Starting test with row_length of %d\n", i);
		// start the timer
		auto start = std::chrono::high_resolution_clock::now();  // Start clock

		test(i);

		// stop the timer
		auto stop = std::chrono::high_resolution_clock::now();  // Stopping the clock
		auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

		printf("Time for learning over %d epochs: %f seconds\n", EPOCHS, duration.count() / 1e6);
		duration_table[i/100 -1][1] = duration.count() / 1e6;
	}

	printf("ROW_LENGTH, TIME\n");
	for (int i = 0; i < (END-START) / STEP_SIZE + 1; i++)
	{
		printf("%.3f, %.3f\n", duration_table[i][0], duration_table[i][1]);
	}

	printf("DONE");

	return 0;
}
