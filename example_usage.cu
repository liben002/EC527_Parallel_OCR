#include "hip/hip_runtime.h"
#include "nn.hpp"
#include <iostream>
#include <time.h>
#include <cassert>

#define EPOCHS 100

// int clock_gettime(clockid_t clk_id, struct timespec *tp);

// double interval(struct timespec start, struct timespec end)
// {
// 	struct timespec temp;
// 	temp.tv_sec = end.tv_sec - start.tv_sec;
// 	temp.tv_nsec = end.tv_nsec - start.tv_nsec;
// 	if (temp.tv_nsec < 0)
// 	{
// 		temp.tv_sec = temp.tv_sec - 1;
// 		temp.tv_nsec = temp.tv_nsec + 1000000000;
// 	}
// 	return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
// }

/**
 * Function to test neural network
 * @returns none
 */
static void test() {
	// Creating network with 3 layers for "iris.csv"
	// First layer neurons must match testing params
	machine_learning::neural_network::NeuralNetwork myNN = machine_learning::neural_network::NeuralNetwork({ {4, "none"}, {60, "relu"}, {30, "sigmoid"} });

	// Printing summary of model
	myNN.summary();

	// Training Model
	myNN.fit_from_csv("iris.csv", true, EPOCHS, 0.3, false, 2, 32, true);

	printf("Testing predictions\n");
	// Testing predictions of model
	assert(machine_learning::argmax(myNN.single_predict({{5, 3.4, 1.6, 0.4}})) == 0);
	assert(machine_learning::argmax( myNN.single_predict({{6.4, 2.9, 4.3, 1.3}})) == 1);
	assert(machine_learning::argmax(myNN.single_predict({{6.2, 3.4, 5.4, 2.3}})) == 2);
	return;
}

/**
 * @brief Main function
 * @returns 0 on exit
 */
int main() {

	struct timespec time_start_CPU, time_end_CPU;

	// start the timer
	auto start = std::chrono::high_resolution_clock::now();  // Start clock

	test();

	// stop the timer
	auto stop = std::chrono::high_resolution_clock::now();  // Stoping the clock
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

	printf("Time for learning over %d epochs: %f seconds\n", EPOCHS, duration.count() / 1e6);
	return 0;
}
