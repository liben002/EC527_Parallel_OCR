#include "hip/hip_runtime.h"
#include "nn.hpp"
#include <iostream>
#include <time.h>
#include <cassert>
#include <omp.h>

#define EPOCHS 100
#define THREADS 4

void detect_threads_setting()
{
	long int i, ognt;
	char * env_ONT;

	//Find out how many threads OpenMP thinks it is wants to use
	#pragma omp parallel for
	for (i=0; i<1; i++) {
		ognt = omp_get_num_threads();
	}

	printf("omp's default number of threads is %d\n", ognt);

	//If this is illegal (0 or less), default to the "#define THREADS" value that is defined above
	if (ognt <= 0) {
		if (THREADS != ognt) {
			printf("Overriding with #define THREADS value %d\n", THREADS);
			ognt = THREADS;
		}
	}

	omp_set_num_threads(ognt);

	// Once again ask OpenMP how many threads it is going to use
	#pragma omp parallel for
	for (i=0; i<1; i++) {
		ognt = omp_get_num_threads();
	}

	printf("Using %d threads for OpenMP\n", ognt);
}

/**
 * Function to test neural network
 * @returns none
 */
static void test() {
	// Creating network with 3 layers for "iris.csv"
	// First layer neurons must match testing params
	machine_learning::neural_network::NeuralNetwork myNN = machine_learning::neural_network::NeuralNetwork({ {4, "none"}, {1200, "relu"}, {1200, "sigmoid"} });

	// Printing summary of model
	myNN.summary();

	// Training Model
	myNN.fit_from_csv("iris.csv", true, EPOCHS, 0.3, false, 2, 32, true);

	printf("Testing predictions\n");
	// Testing predictions of model
	assert(machine_learning::argmax(myNN.single_predict({{5, 3.4, 1.6, 0.4}})) == 0);
	assert(machine_learning::argmax( myNN.single_predict({{6.4, 2.9, 4.3, 1.3}})) == 1);
	assert(machine_learning::argmax(myNN.single_predict({{6.2, 3.4, 5.4, 2.3}})) == 2);
	return;
}

/**
 * @brief Main function
 * @returns 0 on exit
 */
int main() {

	detect_threads_setting();

	// start the timer
	auto start = std::chrono::high_resolution_clock::now();  // Start clock

	test();

	// stop the timer
	auto stop = std::chrono::high_resolution_clock::now();  // Stoping the clock
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

	printf("Time for learning over %d epochs: %f seconds\n", EPOCHS, duration.count() / 1e6);
	return 0;
}
